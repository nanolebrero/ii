
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 1024 
#define ESCRIBE 1
__global__  void avance( float* X, float* V, float* F, const float* Fext, float* salida,const int nodos, const float* mt, const float* fr , bool* tococ )
{
      
    // Block index
    uint num_cuer = blockIdx.x;
    uint nodo_en=threadIdx.x; 
//     printf("num_cuer %d, \n", num_cuer);
    // Thread index
      uint nodo = threadIdx.x+num_cuer*nodos;
      float vel=V[nodo] ;  
      float xr = X[nodo];
      float mtt=mt[nodo_en]/(num_cuer+1.);
      float fext=Fext[nodo];
      bool toco=tococ[num_cuer];
      float frr=1.-fr[nodo_en];

        fext=fext*(0.8 + 0.2/mtt);
//        frr=frr*mtt;
//       printf("nodo,mttyfrr %d % , 
     for (uint ii=0 ; ii < 256*ESCRIBE ; ++ii) { 
        if(nodo_en > 0 && nodo_en < nodos) {
           float fuerza = (X[nodo-1] + X[nodo+1] - 2.*xr);  
           if(toco /*&& ii < 200*/) {

           fuerza += fext ;
//           printf("toco, %d %e \n ", nodo_en ,fext); 
          }
             
           vel += fuerza*mtt ;

           vel = vel*frr;

           xr = xr + vel ;

           X[nodo] = xr;

//         if(nodo_en == 1*nodos/4) printf(" bla %e \n",xr) ;
}
         __syncthreads();
    if(nodo_en == 0 || nodo_en==nodos)xr=0 ;
 
          if (nodo_en == 1*nodos/7) {
           if( ii % ESCRIBE == 0) {
           salida[ii/ESCRIBE+num_cuer*256]=xr ;//sale/ESCRIBE;
          }
          }
   }
/*         if (nodo_en == 1*nodos/4) {
          for(uint kk=0; kk < 256 ; ++kk) {

            printf("sale %d \n", salida[kk]) ;
           }
           }*/

           V[nodo] = vel ;

             

}


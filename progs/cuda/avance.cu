
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 1024 
#define ESCRIBE 2
__global__  void avance( float* X, float* V, float* F, const float* Fext, float* salida,const int nodos, const float* mt, const float* fr , bool* tococ, const float random )
{
    // Block index
    uint num_cuer = blockIdx.x;
    uint nodo_en=threadIdx.x; 
//     printf("num_cuer %d, \n", num_cuer);
    // Thread index
      uint nodo = threadIdx.x+num_cuer*nodos;
      float vel=V[nodo] ;  
      float xr = X[nodo];
      float mtt=mt[nodo_en]/(num_cuer+1.);
      float fext=Fext[nodo];//*random;
      bool toco=tococ[num_cuer];
      float frr=1.-fr[nodo_en];
      float fuerza_vieja=0. ;
       bool suelta ;
        fext=fext*(0.8 + 0.2/mtt);
//        frr=frr*mtt;
//       printf("nodo,mttyfrr %d % , 
     for (uint ii=0 ; ii < 256*ESCRIBE ; ++ii) { 
        if(nodo_en > 0 && nodo_en < nodos) {
               float fuerza ; 
          fuerza = (X[nodo-1] + X[nodo+1] - 2.*xr);
        // if(nodo_en > 2 && nodo_en < nodos-2)  fuerza = (X[nodo-3] + X[nodo+3] - 2.*xr);
        // if(nodo_en <= 2) fuerza = (0. + X[nodo+3] - 2.*xr); 
        // if(nodo_en >= nodos-2) fuerza = (0. + X[nodo-3] - 2.*xr); 

        //         fuerza +=-300*(pow((xr-X[nodo-1]),3) + pow((xr-X[nodo+1]),3)) ;
//           float fuerza = (X[nodo-1] + X[nodo+1] - 2.00001*xr); // PARECE ALGO 3D
           if(toco && ii < 2000) {

            //if(vel >= 0.02*random) fuerza += fext ;
             fuerza+= fext*10;//(-0.1*ii*ii+51.2*ii) ;
             // frr = 0.8;
         }
           
           vel += fuerza*mtt ;
           vel = vel*frr;
           xr = xr + vel ;
           X[nodo] = xr;
}
         __syncthreads();
    if(nodo_en == 0 || nodo_en==nodos) { 
             xr=0 ;
             X[nodo]=xr ;
          }
          if (nodo_en == 1*nodos/7) {
           if( ii % ESCRIBE == 0) {
           salida[ii/ESCRIBE+num_cuer*256]=xr ;//sale/ESCRIBE;
          }
          }
   }
           V[nodo] = vel ;
}



#include <hip/hip_runtime.h>
#define BLOCK_SIZE 1024 
#define ESCRIBE 1
__global__  void avance( float* X, float* V, float* F, const float* Fext, float* salida,const int nodos, const float* mt, const float* fr , bool toco )
{
      
    // Block index
    int num_cuer = blockIdx.x;
//     printf("Hola HOla %d \n", num_cuer) ;
      int nodo_en=threadIdx.x; 
    // Thread index
      int nodo = threadIdx.x+num_cuer*nodos;
      float vel=V[nodo] ;  
      float xr = X[nodo];
      float mtt=mt[nodo_en]/(0.2*num_cuer+1);
      float fext=Fext[nodo_en];
//       if(num_cuer==17)  xr=0;
//       if(num_cuer==0)     mtt=mt[nodo_en];
      float frr=1.-fr[nodo_en];
//     printf("frr %e \n", frr) ;

    for (int ii=0 ; ii < 256*ESCRIBE ; ++ii) { 
     if(nodo_en > 0 && nodo_en < nodos) {
     float fuerza = (X[nodo-1] + X[nodo+1] - 2.*xr);  
     if(toco && ii < 200) fuerza += fext ;

        vel += fuerza*mtt ;

        vel= vel*frr;

         xr = xr + vel ;

          X[nodo]=xr;
}
/*    if(num_cuer < 5) {
    if(nodo_en==nodos/2) {
     xr=0;

     }
     }*/
         __syncthreads();
    if(nodo_en == 0 || nodo_en==nodos)xr=0 ;
                             
 
          if (nodo_en == 7*nodos/12) {
           if( ii % ESCRIBE == 0) {
           salida[ii/ESCRIBE+num_cuer*256]=xr ;//sale/ESCRIBE;
//           printf("Hola HOla %e %d %d \n", salida[ii/ESCRIBE+num_cuer*256], num_cuer, nodos) ;
          }
          }
   }

           V[nodo] = vel ;

             

}


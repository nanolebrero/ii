#include "hip/hip_runtime.h"
//#include <hipblas.h> 
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "avance.cu"
 
// Allocates a matrix with random float entries.
void ceroInit(float* data, int size)
{
   for (int i = 0; i < size; ++i)
   data[i] = 0.;
}
void gaussiana(float* data,  int nodos, int cuerdas)
{
       float cosa = 0;
     for(int i= 0; i< cuerdas; ++i){ 
     for(int j= 0; j < nodos; ++j){ 
      cosa = -0.02*((j-1*nodos/3)*(j-1*nodos/3)) ;     
     data[j+i*nodos] = 2.E-3*exp(cosa);
    // data[j] =( cosa-cosa*cosa/1024.);//^2/1024;
 //   printf ("data %e  \n",   data[j+i*nodos]) ;
   }
  }
}
void randommasa(float* data, int nodos,int cuerdas, float masa, float orden)
{
  for (int j=0; j< cuerdas; ++j) {
   for (int i = 0; i < nodos; ++i) {
 
   data[i+j*nodos] = masa - ((float) rand()/RAND_MAX)*orden*masa;
//    printf ("data %e  \n",   data[i]) ;
    }
  }
}
void friccionpuntas(float* data, int nodos,int cuerdas, float param, float orden)
{
   for (int j=0; j < cuerdas; ++j){
   for (int i = 0; i < nodos; ++i) {
   float factor = 0.2*exp(-orden*(i*i)) + exp(-orden*(i-nodos)*(i-nodos));
   data[i+j*nodos] += factor;
//    printf ("data %e  \n",   data[i]) ;
    }
  }
}
void random(float* data, int nodos,int cuerdas, int i)
{
        //  float cosa = 0;
     for(int j= 0; j < nodos; ++j){
       float cosa=data[j+i*nodos];
     data[j+i*nodos] = cosa*((float) rand()/RAND_MAX-0.5);
  }
}
 
/////////////////////////////////////////////////////////
// Program main
/////////////////////////////////////////////////////////
 
int
main(int argc, char** argv)
{

   // set seed for rand()
   //srand(2006);

 int npart=atoi(argv[1]);    // Cantidad de nodos en cada cuerda	
 int ncuerdas=atoi(argv[2]);  // cantidad de cuerdas
 int npasos=atoi(argv[3]);   // Cantidad de pasos
 float masatension=  atof(argv[4]); // Factor masa-tension
 float friccion=  atof(argv[5]);    // factor de fricción
 bool toco=false ;
   // 1. allocate host memory for matrices A and B
   unsigned int size_A = npart*ncuerdas;
   unsigned int mem_size_A = sizeof(float) * size_A;
   unsigned int buffer_salida = sizeof(float)*256*ncuerdas;
   unsigned int bcur = sizeof(bool)*ncuerdas;   

   float* X = (float*) malloc(mem_size_A);
   float* V = (float*) malloc(mem_size_A);
   float* F = (float*) malloc(mem_size_A);
   float* Fr = (float*) malloc(mem_size_A);
   float* M = (float*) malloc(mem_size_A);
   float* Fext = (float*) malloc(mem_size_A);
   float* salida=(float*) malloc(buffer_salida);
   bool* tococ=(bool*) malloc(bcur);
 
   // 8. allocate device memory
   float* d_X;
   float* d_V;
   float* d_F;
   float* d_Fr;
   float* d_M;
   float* d_Fext;
   float* d_salida;
   bool* d_tococ;
   hipMalloc((void**) &d_X, mem_size_A);
   hipMalloc((void**) &d_V, mem_size_A);
   hipMalloc((void**) &d_F, mem_size_A);
   hipMalloc((void**) &d_Fr, mem_size_A);
   hipMalloc((void**) &d_M, mem_size_A);
   hipMalloc((void**) &d_Fext, mem_size_A);
   hipMalloc((void**) &d_salida, buffer_salida);
   hipMalloc((void**) &d_tococ, bcur);

//    hipblasHandle_t handle;
//    hipblasCreate(&handle);

   // 2. initialize host memory
    ceroInit(X, size_A);
    ceroInit(V, size_A);
    ceroInit(F, size_A);
//    gaussiana(X, size_A);
    gaussiana(Fext, npart, ncuerdas);
    randommasa(M, npart, ncuerdas, masatension, 0.1);
    randommasa(Fr, npart, ncuerdas, friccion, 0.0);
    friccionpuntas(Fr, npart, ncuerdas, friccion, .10);
   

   // 9. copy host memory to device

   hipMemcpy(d_X, X, mem_size_A, 
   hipMemcpyHostToDevice);
   hipMemcpy(d_V, V, mem_size_A, 
   hipMemcpyHostToDevice);
   hipMemcpy(d_F, F, mem_size_A, 
   hipMemcpyHostToDevice);
   hipMemcpy(d_Fr, Fr, mem_size_A, 
   hipMemcpyHostToDevice);
   hipMemcpy(d_M, M, mem_size_A, 
   hipMemcpyHostToDevice);
   hipMemcpy(d_Fext, Fext, mem_size_A, 
   hipMemcpyHostToDevice);

 /*  for (uint nn=0; nn<npart; ++nn) {

    printf ("masa %e  \n", M[nn]) ;
    printf ("friccion %e  \n", Fr[nn]) ;

    } 
*/
    int sig=1;
   for (uint i = 0 ; i < npasos ; ++i) 
 { 
   
 
   // 5. perform the calculation
   // setup execution parameters
   dim3 threads(npart);
   dim3 grid(ncuerdas);
   // execute the kernel

  float random;
  for(int ii=0; ii < ncuerdas ; ++ii) {   

     tococ[ii] = 0;

      
 //    if(i > ii*450 + 10 && i < (ii + 1)*450 +20 ) {
     if(i == ii*400 + 10 ) { //&& i < (ii + 1)*150 +20 ) {
         tococ[ii]=1;
//      printf("toco la cuerda, %d %d", ii, i) ;
//      random(Fext, npart, ncuerdas,ii);
      }
    }
//   hipMemcpy(d_Fext, Fext, mem_size_A, 
//   hipMemcpyHostToDevice);

   hipMemcpy(d_tococ, tococ, bcur, 
   hipMemcpyHostToDevice);

/*    if(i % 100 == 0 ) {

       sig=-1*sig;
      }*/
       random=sig*((float) rand()/RAND_MAX-0.5);
   

   avance<<< grid, threads >>>(d_X, d_V, 
                                  d_F, d_Fext, d_salida, npart, d_M, d_Fr, d_tococ, random);
   

   // 11. copy result from device to host
   hipMemcpy(salida, d_salida, buffer_salida, 
   hipMemcpyDeviceToHost);
/*    for (uint j = 0 ; j< npart; ++j) {
    printf ("Xs %e  %d \n", X[j], j ) ; 
             }
*/
    for (uint j = 0 ; j< 256; ++j) {
       int sal =0 ;
       int sal2 =0 ;
         for (uint i=0; i< ncuerdas  ;++i) {
        if(((i+1) % 2) == 0){
         sal += 100000*salida[j+i*256];
          }
          else {
         sal2 += 100000*salida[j+i*256];
             }
                  }
    printf (" %d , %d  \n", sal , sal2) ; 
            }
   }
 
}


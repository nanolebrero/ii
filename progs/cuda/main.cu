#include "hip/hip_runtime.h"

// Multiply two matrices A * B = C
//#include <hipblas.h> 
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include "avance.cu"
 
// Allocates a matrix with random float entries.
void ceroInit(float* data, int size)
{
   for (int i = 0; i < size; ++i)
   data[i] = 0.;
}
void gaussiana(float* data,  int nodos, int cuerdas)
{
       float cosa = 0;
     for(int i= 0; i< cuerdas; ++i){ 
     for(int j= 0; j < nodos; ++j){ 
      cosa = -0.01*((j-2*nodos/10)*(j-2*nodos/10)) ;     
     data[j+i*nodos] = 2.E-3*exp(cosa);
    // data[j] =( cosa-cosa*cosa/1024.);//^2/1024;
//    printf ("data %e  \n",   cosa) ;
   }
  }
}
void randommasa(float* data, int nodos,int cuerdas, float masa, float orden)
{
  for (int j=0; j< cuerdas; ++j) {
   for (int i = 0; i < nodos; ++i) {
 
   data[i+j*nodos] = masa - ((float) rand()/RAND_MAX)*orden*masa;
//    printf ("data %e  \n",   data[i]) ;
    }
  }
}
void friccionpuntas(float* data, int nodos,int cuerdas, float param, float orden)
{
   for (int j=0; j < cuerdas; ++j){
   for (int i = 0; i < nodos; ++i) {
   float factor = exp(-orden*(i*i)) + exp(-orden*(i-nodos)*(i-nodos));
   data[i+j*nodos] += factor;
//    printf ("data %e  \n",   data[i]) ;
    }
  }
}
 
/////////////////////////////////////////////////////////
// Program main
/////////////////////////////////////////////////////////
 
int
main(int argc, char** argv)
{

   // set seed for rand()
   //srand(2006);

 int npart=atoi(argv[1]);    // Cantidad de nodos en cada cuerda	
 int ncuerdas=atoi(argv[2]);  // cantidad de cuerdas
 int npasos=atoi(argv[3]);   // Cantidad de pasos
 float masatension=  atof(argv[4]); // Factor masa-tension
 float friccion=  atof(argv[5]);    // factor de fricción
 bool toco=false ;
   // 1. allocate host memory for matrices A and B
   unsigned int size_A = npart*ncuerdas;
   unsigned int mem_size_A = sizeof(float) * size_A;
   unsigned int buffer_salida = sizeof(float)*256*ncuerdas;

   float* X = (float*) malloc(mem_size_A);
   float* V = (float*) malloc(mem_size_A);
   float* F = (float*) malloc(mem_size_A);
   float* Fr = (float*) malloc(mem_size_A);
   float* M = (float*) malloc(mem_size_A);
   float* Fext = (float*) malloc(mem_size_A);
   float* salida=(float*) malloc(buffer_salida);

 
   // 8. allocate device memory
   float* d_X;
   float* d_V;
   float* d_F;
   float* d_Fr;
   float* d_M;
   float* d_Fext;
   float* d_salida;
   hipMalloc((void**) &d_X, mem_size_A);
   hipMalloc((void**) &d_V, mem_size_A);
   hipMalloc((void**) &d_F, mem_size_A);
   hipMalloc((void**) &d_Fr, mem_size_A);
   hipMalloc((void**) &d_M, mem_size_A);
   hipMalloc((void**) &d_Fext, mem_size_A);
   hipMalloc((void**) &d_salida, buffer_salida);

//    hipblasHandle_t handle;
//    hipblasCreate(&handle);

   // 2. initialize host memory
    ceroInit(X, size_A);
    ceroInit(V, size_A);
    ceroInit(F, size_A);
//    gaussiana(X, size_A);
    gaussiana(Fext, npart, ncuerdas);
    randommasa(M, npart, ncuerdas, masatension, 0.1);
    randommasa(Fr, npart, ncuerdas, friccion, 0.1);
    friccionpuntas(Fr, npart, ncuerdas, friccion, 0.1);
   

   // 9. copy host memory to device

   hipMemcpy(d_X, X, mem_size_A, 
   hipMemcpyHostToDevice);
   hipMemcpy(d_V, V, mem_size_A, 
   hipMemcpyHostToDevice);
   hipMemcpy(d_F, F, mem_size_A, 
   hipMemcpyHostToDevice);
   hipMemcpy(d_Fr, Fr, mem_size_A, 
   hipMemcpyHostToDevice);
   hipMemcpy(d_M, M, mem_size_A, 
   hipMemcpyHostToDevice);
   hipMemcpy(d_Fext, Fext, mem_size_A, 
   hipMemcpyHostToDevice);


   for (uint i = 0 ; i < npasos ; ++i) 
 { 
   
 
   // 5. perform the calculation
   // setup execution parameters
   dim3 threads(npart);
   dim3 grid(ncuerdas);
//     printf("pancho %e %e \n", masatension, friccion) ; 
   // execute the kernel
   if(10 < i && i < 12) {
   avance<<< grid, threads >>>(d_X, d_V, 
                                  d_F, d_Fext, d_salida, npart, d_M, d_Fr, 1);
 } else {
   avance<<< grid, threads >>>(d_X, d_V, 
                                 d_F, d_Fext, d_salida, npart, d_M, d_Fr, 0);

}
   


   // 11. copy result from device to host
   hipMemcpy(salida, d_salida, buffer_salida, 
   hipMemcpyDeviceToHost);
//   hipMemcpy(X, d_X, mem_size_A, 
//   hipMemcpyDeviceToHost);
/*    for (uint j = 0 ; j< npart; ++j) {
    printf ("Xs %e  %d \n", X[j], j ) ; 
             }
*/
    for (uint j = 0 ; j< 256; ++j) {
       int sal =0 ;
       int sal2 =0 ;
         for (uint i=0; i< ncuerdas/2 ;++i) {
         sal += 100000*salida[j+i*256];
         sal2 += 100000*salida[j+i*256*2];
                  }
    printf ("  %d , %d  \n", sal , sal2) ; 
            }
   }
 
}

